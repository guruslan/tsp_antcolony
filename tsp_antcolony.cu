#include "hip/hip_runtime.h"
/*
 * Copyright 2010 Ruslan Kudubayev.
 */

/* 
 * Host code.
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <cutil_inline.h>

// includes, kernels
#include <MersenneTwister_kernel.cu>
#include <tsp_antcolony_kernel.cu>


void printArr(float *data1, int size)
{
  int i,j,k;
  for (j=0; j<size; j++) {
    for (i=0; i<size; i++) {
      k = j*size+i;
      printf("%1.20f ", data1[k]);
    }
    printf("\n");
  }
}

FILE *infile;

int getSize(char* filename)
{
	char line[100];
	int size;
	/* Open the file.  If NULL is returned there was an error */
	if((infile = fopen(filename, "r")) == NULL) {
		printf("Error Opening File.\n");
		exit(1);
	}
	fgets(line, sizeof(line), infile);
	printf(line);
	fgets(line, sizeof(line), infile);
	printf(line);
	fgets(line, sizeof(line), infile);
	printf(line);
	fscanf(infile, "DIMENSION : %d\n", &size);
	printf("%d\n", size);
	fgets(line, sizeof(line), infile);
	printf(line);
	fgets(line, sizeof(line), infile);
	printf(line);
	fgets(line, sizeof(line), infile);
	printf(line);
	return size;
}

void getGraph(int* data, int size) {
	for (int i = 0; i < size*size; ++i) {
        fscanf(infile,"%d", &data[i]);
    }
    fclose(infile);  /* Close the file */
}

void outputfordotformatf(char* filename, int* delta, float* data, int* path, int size, float scale)
{
	FILE *outfile;
	/* Open the file.  If NULL is returned there was an error */
	if((outfile = fopen(filename, "w")) == NULL) {
		printf("Error Opening File.\n");
		exit(1);
	}
	
	fputs("digraph G{\n",outfile);
	
	for (int i = 0; i < size; ++i) {
		for (int j = 0; j < size; ++j) if (i!=j && abs(data[i*size+j])>=1e-16 && delta[i*size+j]<1000000) {
			if (path[i] == j) {
        		fprintf(outfile,"	%d -> %d [label=\"%f\", penwidth=%f, color=red];\n", i, j, data[i*size+j], data[i*size+j]*scale);
			} else {
        		fprintf(outfile,"	%d -> %d [label=\"%f\", penwidth=%f];\n", i, j, data[i*size+j], data[i*size+j]*scale);
        	}
        }
    }
	fputs("}",outfile);
    
    fclose(outfile);  /* Close the file */

}

void outputfordotformati(char* filename, int* data, int* path, int size, float scale)
{
	FILE *outfile;
	/* Open the file.  If NULL is returned there was an error */
	if((outfile = fopen(filename, "w")) == NULL) {
		printf("Error Opening File.\n");
		exit(1);
	}
	
	fputs("digraph G{\n",outfile);
	
	for (int i = 0; i < size; ++i) {
		for (int j = 0; j < size; ++j) if (i!=j && data[i*size+j]>0 && data[i*size+j]<1000000) {
			if (path[i] == j) {
        		fprintf(outfile,"	%d -> %d [label=\"%d\", penwidth=%f, color=red];\n", i, j, data[i*size+j], data[i*size+j]*scale);
			} else {
        		fprintf(outfile,"	%d -> %d [label=\"%d\", penwidth=%f];\n", i, j, data[i*size+j], data[i*size+j]*scale);
        	}
        }
    }
	fputs("}",outfile);
    
    fclose(outfile);  /* Close the file */
}

//performs a nearest neighbour simple greedy search to get a value for tau0.
float nearest_neighbour(int* h_A, int size) {
	int cur = 0;
	int visited[WA];
	float res = 0;
	for (int i=0; i<size; i++) visited[i] = 0;
	visited[cur] = 1;
	for (int i=1; i<size; i++) {
		int min = 214748364;
		int minj = i;
		for (int j=0; j<size; j++) if (visited[j] == 0) {
			if (h_A[size*cur+j] < min) {
				min = h_A[size*cur+j];
				minj = j;
			}
		}
		res += min;
		visited[minj] = 1;
		cur = minj;
	}
	res += h_A[size*cur + 0];
	return res;
}

//Align a to nearest higher multiple of b
extern "C" int iAlignUp(int a, int b){
    return ((a % b) != 0) ?  (a - a % b + b) : a;
}

//ceil(a / b)
extern "C" int iDivUp(int a, int b){
    return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

extern "C" void initMTRef(const char *fname);

////////////////////////////////////////////////////////////////////////////////
// Run test
////////////////////////////////////////////////////////////////////////////////
void runTest(int argc, char** argv)
{
	char* filenametotest;
	int targetanswer;
	if (argc > 2) {
		// parameters to the executable:
		// 1. filename
		// 2. the answer to the problem as an integer 
		filenametotest = argv[1];
		targetanswer = atoi(argv[2]);
	}
	// get the size of the problem from the file first
    const int n = getSize(filenametotest);
    
    // dividing the number of ants evenly into blocks for computation
    int blocks = iDivUp(ANTS, BLOCK_SIZE);
    printf("Blocks: %d\n", blocks);
        
    // allocate host memory for matrice A
    unsigned int size_A = n * n;
    unsigned int mem_size_A = sizeof(int) * size_A;
    int* h_A = (int*) malloc(mem_size_A);
    getGraph(h_A, n);
    
    hipSetDevice( cutGetMaxGflopsDeviceId() );

    // set seed for rand()
    srand((unsigned)time(NULL));
    
		/*
		This routine is responsible for preparing to generating random numbers on the device and leaving
		them there for the other kernel to use.
		*/
   		float *d_Rand;
   		
   		int path_n = (2*n-1) * BLOCK_SIZE * blocks;
   		
		int n_per_rng = iAlignUp(iDivUp(path_n, MT_RNG_COUNT), 2);
		int rand_n = MT_RNG_COUNT * n_per_rng;
	
    	//printf("Initializing data on the device for %i random samples...\n", path_n);
        cutilSafeCall( hipMalloc((void **)&d_Rand, rand_n * sizeof(float)) );
        
        initMTRef("MersenneTwister.raw");
        loadMTGPU("MersenneTwister.dat");
        

    // allocate device memory for storing delta matrix
    int* d_A;
    cutilSafeCall(hipMalloc((void**) &d_A, mem_size_A));
    // copy host memory to device
    cutilSafeCall(hipMemcpy(d_A, h_A, mem_size_A,
                              hipMemcpyHostToDevice) );

    // allocate device memory for tau matrix
    unsigned int size_C = n * n;
    unsigned int mem_size_C = sizeof(float) * size_C;
    float* d_C;
    cutilSafeCall(hipMalloc((void**) &d_C, mem_size_C));
    // allocate host memory for the tau on host
    float* h_C = (float*) malloc(mem_size_C);
    //initialise pheromones tau matrix
    float tau0 = 1.0f/((float)n * nearest_neighbour(h_A, n));  
    //printf("tau0: %1.20f\n", tau0);
    for (int i=0; i<size_C; ++i) h_C[i] = tau0;
    // copy host memory to device
    cutilSafeCall(hipMemcpy(d_C, h_C, mem_size_C,
                              hipMemcpyHostToDevice) );
    
    // allocate device memory for the path vector
    int size_P = n;
    int mem_size_P = sizeof(int) * size_P;
    int* d_P;
    cutilSafeCall(hipMalloc((void**) &d_P, mem_size_P));
    // allocate host memory for the R on host
    int* h_P = (int*) malloc(mem_size_P);
    // copy host memory to device
    cutilSafeCall(hipMemcpy(d_P, h_P, mem_size_P,
                              hipMemcpyHostToDevice) );
    
    // allocate device memory for best on the iteration
    int* d_best;
    cutilSafeCall(hipMalloc((void**) &d_best, sizeof(int)));
    // allocate host memory for the best on host
    int* h_best = (int*)malloc(sizeof(int));
    *h_best = 2147483647;
    int global_best = 2147483647;
    // copy host memory to device
    cutilSafeCall(hipMemcpy(d_best, h_best, sizeof(int),
                              hipMemcpyHostToDevice) ); 

    // create and start timer
    unsigned int timer = 0;
    cutilCheckError(cutCreateTimer(&timer));
    cutilCheckError(cutStartTimer(timer));
    
    // dimensions of the global update kernel
    dim3 threads2(BLOCK_SIDE_UPDATER, BLOCK_SIDE_UPDATER);
    int side_blocks = n/BLOCK_SIDE_UPDATER;
    if (n%BLOCK_SIDE_UPDATER != 0) side_blocks++;
    dim3 grid2(side_blocks, side_blocks);

	// *****************************************************
	// outputfordotformatf("tau_graph0.dot",h_A,h_C,h_P,n,500.0f);
	// the main block of code that executes the kernel.
	int firsttimeto20 = 0;
	for (int iteration=0; iteration<2048; ++iteration) {
		//generate random numbers for this iteration
		seedMTGPU(rand()%100000);
		RandomGPU<<<32, 128>>>(d_Rand, n_per_rng);
        cutilCheckMsg("RandomGPU() execution failed\n");
        cutilSafeCall( hipDeviceSynchronize() );
	
		// this is actually the rho and phi constants that are just taken to be the same value.
		float damping = 0.1f;
		
    	// execute the kernel
    	colonise<<< blocks, BLOCK_SIZE >>>(d_C, d_A, d_Rand, d_best, d_P, n, damping, tau0);
        cutilSafeCall( hipDeviceSynchronize() );
          	
    	// get the best so far
	    cutilSafeCall(hipMemcpy(h_best, d_best, sizeof(int),
                              hipMemcpyDeviceToHost) );
    	cutilSafeCall(hipMemcpy(h_P, d_P, mem_size_P,
                              hipMemcpyDeviceToHost) );
	    cutilSafeCall(hipMemcpy(h_C, d_C, mem_size_C,
                              hipMemcpyDeviceToHost) );
            
            
        /*
        if (iteration==0 || iteration==1 || iteration==2 || iteration==3 || iteration==5 || iteration==6 || iteration==10 || iteration==15 || iteration==20 || iteration==25 || iteration==30 || iteration==100 || iteration==500 || iteration==1000 || iteration==2000) {
        	char name[100] = "tau_graphN.dot";
        	sprintf(name, "tau_graph%d.dot", iteration+1);
        	outputfordotformatf(name,h_A,h_C,h_P,n,500.0f);
        }
        */
        
        // record when we get to 20% accuracy
        if ((firsttimeto20 == 0) && (((float)*h_best/targetanswer) <= 1.2f)) {
        	firsttimeto20 = 1;
        	printf("First time to 20 percent accuracy: %f (ms) \n", cutGetTimerValue(timer));
        	printf("Convergenece interation: %d\n",iteration+1);
        }
        // if reached the optimal answer then quit, no point to work anymore.
    	//if (*h_best == targetanswer) break;
    	
    	// global updating rule here.
    	// can just execute another kernel here which would do that.
    	// the reason is to not copy all the data forth and back but do modifications over there.
    	update_pheromones<<< grid2, threads2 >>>(d_C, d_best, d_P, n, damping);
        cutilSafeCall( hipDeviceSynchronize() );

        if (*h_best < global_best) {
        	global_best = *h_best;
        }
  		*h_best = 2147483647;
    	cutilSafeCall(hipMemcpy(d_best, h_best, sizeof(int),
                              hipMemcpyHostToDevice) ); 
    }
    // ******************************************************  

    // stop and destroy timer
    cutilCheckError(cutStopTimer(timer));
    printf("Processing time: %f (ms) \n", cutGetTimerValue(timer));
    
    cutilCheckError(cutDeleteTimer(timer));

    // check if kernel execution generated and error
    cutilCheckMsg("Kernel execution failed"); 

    // copy result from device to host
    cutilSafeCall(hipMemcpy(h_C, d_C, mem_size_C,
                              hipMemcpyDeviceToHost) );
    cutilSafeCall(hipMemcpy(h_P, d_P, mem_size_P,
                              hipMemcpyDeviceToHost) );
    cutilSafeCall(hipMemcpy(h_best, d_best, sizeof(int),
                              hipMemcpyDeviceToHost) );

    printf("Result: %d\n", global_best);

    //printf("Tau:\n");
    //printArr(h_C,n);
    
    //outputfordotformati("original_graph.dot",h_A,h_P,n,0.1f);
    outputfordotformatf("tau_graph.dot",h_A,h_C,h_P,n,1200.0f);

    // clean up memory
    free(h_A);
    free(h_C);
    free(h_P);
    free(h_best);
    cutilSafeCall(hipFree(d_A));
    cutilSafeCall(hipFree(d_C));
    cutilSafeCall(hipFree(d_Rand));
    cutilSafeCall(hipFree(d_P));
    cutilSafeCall(hipFree(d_best));

    hipDeviceReset();
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv)
{
    runTest(argc, argv);
    cutilExit(argc, argv);
}

